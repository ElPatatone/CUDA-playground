
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

void vectorAddCPU(float* x, float* y, float* z, int N) {
    for (int i = 0; i < N; i++) {
        z[i] = x[i] + y[i];
    }
}

void vectorAddGPU(float* x, float* y, float* z, int N) {
    // 1. allocate GPU memory
    float *x_d, *y_d, *z_d;

    hipMalloc((void**)&x_d, sizeof(float)*N);
    hipMalloc((void**)&y_d, sizeof(float)*N);
    hipMalloc((void**)&z_d, sizeof(float)*N);

    // 2. copy data from CPU memory to GPU memory
    hipMemcpy(x_d, x, sizeof(float)*N, hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, sizeof(float)*N, hipMemcpyHostToDevice);
    
    // 3. perform computation on GPU
    
    // 4. copy data from GPU memory to CPU memory
    hipMemcpy(z_d, z, sizeof(float)*N, hipMemcpyDeviceToHost);

    // 5. deallocate GPU memory
    hipFree(&x_d);
    hipFree(&y_d);
    hipFree(&z_d);

}


int main (int argc, char *argv[]) {

    int N = (argc > 1) ? atoi(argv[1]) : (1 << 25);

    float* x = (float*) malloc(sizeof(float)*N);
    float* y = (float*) malloc(sizeof(float)*N);
    float* z = (float*) malloc(sizeof(float)*N);

    for (int i = 0; i < 1; i++) {
        x[i] = rand();
        y[i] = rand();
    }

    vectorAddCPU(x, y, z, N);

    return 0;
}
