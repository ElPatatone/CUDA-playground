
#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>

#define WIDTH 8192
#define HEIGHT 8192
#define IMAGE_SIZE (WIDTH * HEIGHT)

__global__ void rgbTogrey_kernel(unsigned char *r, unsigned char *g, 
                                 unsigned char *b, unsigned char *grey) {

    unsigned int row = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int col = blockDim.x * blockIdx.x + threadIdx.x;

    // gets the individual pixel in the image
    unsigned int i = row*WIDTH + col;
    // boundary checks
    if (col < WIDTH && row < HEIGHT) {
        grey[i] = (unsigned char)(r[i]*0.299 + g[i]*0.587 + b[i]*0.114);
    }
}

void rgbTogrey_cpu(unsigned char *r, unsigned char *g,
                   unsigned char *b, unsigned char *grey) {

    // planar RGB processing
    for (int i = 0; i < IMAGE_SIZE; i++) {
        grey[i] = (unsigned char)(r[i]*0.299 + g[i]*0.587 + b[i]*0.114);
    }
}


void rgbTogrey_gpu(unsigned char *r, unsigned char *g,
                   unsigned char *b, unsigned char *grey) {

    // 1. allocate gpu memory
    unsigned char *r_d, *g_d, *b_d, *grey_d;
    
    hipMalloc((void**)&r_d, sizeof(unsigned char)*IMAGE_SIZE);
    hipMalloc((void**)&g_d, sizeof(unsigned char)*IMAGE_SIZE);
    hipMalloc((void**)&b_d, sizeof(unsigned char)*IMAGE_SIZE);
    hipMalloc((void**)&grey_d, sizeof(unsigned char)*IMAGE_SIZE);

    // 2. copy data from cpu to gpu
    hipMemcpy(r_d, r, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(g_d, g, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyHostToDevice);

    // 3. run the kernel
    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((WIDTH + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x, 
                   (HEIGHT + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    rgbTogrey_kernel<<<numBlocks, numThreadsPerBlock>>>(r_d, g_d, b_d, grey_d);
    hipEventRecord(stop, 0);

    hipError_t err = hipGetLastError();  // Check launch errors immediately
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();  // Wait for kernel to finish
    
    hipError_t error = hipEventSynchronize(stop);
    if (error != hipSuccess) {
        printf("[Error] Kernel failed to launch: %s", hipGetErrorString(error));
    }

    float timeTaken = 0;
    hipEventElapsedTime(&timeTaken, start, stop);
    printf("GPU computation took: %.10f seconds\n", timeTaken/1000);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 4. copy data from cpu to gpu
    hipMemcpy(grey, grey_d, sizeof(unsigned char)*IMAGE_SIZE, hipMemcpyDeviceToHost);

    // 5. free the gpu memory
    hipFree(r_d);
    hipFree(g_d);
    hipFree(b_d);
    hipFree(grey_d);
}

int main(int argc, char *argv[]) {

    // unsigned char rgb[IMAGE_SIZE*3];
    unsigned char *r = (unsigned char*) malloc(IMAGE_SIZE); 
    unsigned char *g = (unsigned char*) malloc(IMAGE_SIZE); 
    unsigned char *b = (unsigned char*) malloc(IMAGE_SIZE); 

    for (int i = 0; i < IMAGE_SIZE; i++) {
        r[i] = i % 256;
        g[i] = (i*2) % 256;
        b[i] = (i*3) % 256;
    }

    // unsigned char r[IMAGE_SIZE] = {255, 0, 0, 255};
    // unsigned char g[IMAGE_SIZE] = {0, 255, 0, 255};
    // unsigned char b[IMAGE_SIZE] ={0, 0, 255, 255};
    unsigned char *grey_cpu = (unsigned char*) malloc(IMAGE_SIZE);
    unsigned char *grey_gpu = (unsigned char*) malloc(IMAGE_SIZE);
    
    rgbTogrey_gpu(r, g, b, grey_gpu);

    clock_t start = clock();
    rgbTogrey_cpu(r, g, b, grey_cpu);
    clock_t stop = clock();

    float timeTaken = (float) (stop - start) / CLOCKS_PER_SEC;
    printf("CPU computation took: %.10f seconds\n", timeTaken);

    for (int i = 0; i < IMAGE_SIZE; i++) {
        if (grey_cpu[i] != grey_gpu[i]) {
            printf("Mismatch in the elements at index: %d\n", i);
            printf("grey_cpu: %d\n", grey_cpu[i]);
            printf("grey_gpu: %d\n", grey_gpu[i]);
        }
    }

    for (int i = 0; i < 10; i++) {
        printf("index: %d\n", i);
        printf("grey_gpu: %d\n", grey_gpu[i]);
        printf("grey_cpu: %d\n", grey_cpu[i]);
    }

    free(r);
    free(g);
    free(b);
    free(grey_cpu);
    free(grey_gpu);

    return 0;
}
